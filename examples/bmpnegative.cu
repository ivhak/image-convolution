#include "hip/hip_runtime.h"
extern "C" {
#include "../lib/bitmap.h"
#include "../src/tools/tools.h"
}
__global__ void negative_kernel(pixel *rawdata_in, pixel *rawdata_out, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    rawdata_out[y*width+x].r = 255 - rawdata_in[y*width+x].r;
    rawdata_out[y*width+x].g = 255 - rawdata_in[y*width+x].g;
    rawdata_out[y*width+x].b = 255 - rawdata_in[y*width+x].b;
}

int main(int argc, char *argv[]) {

    char *filename_in  = argv[1];
    char *filename_out = argv[2];

    image_t *in = image_from_filename(filename_in);
    image_t *out = new_image(in->width, in->height);

    const int width  = in->width;
    const int height = in->height;

    const int size_of_all_pixels = width*height*sizeof(pixel);

    pixel *d_rawdata_in, *d_rawdata_out;

    hipMalloc((void **)&d_rawdata_in,  size_of_all_pixels);
    hipMalloc((void **)&d_rawdata_out, size_of_all_pixels);

    hipMemcpy(d_rawdata_in, in->rawdata, size_of_all_pixels, hipMemcpyHostToDevice);

    dim3 block_size(32, 32);
    dim3 grid_size((width  + block_size.x - 1) / block_size.x,
                   (height + block_size.y - 1) / block_size.y);

    negative_kernel<<<grid_size, block_size>>>(d_rawdata_in, d_rawdata_out, width, height);


    hipMemcpy(out->rawdata, d_rawdata_out, size_of_all_pixels, hipMemcpyDeviceToHost);

    save_image(out, filename_out);
}
